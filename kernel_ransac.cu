
#include <hip/hip_runtime.h>
#include "float.h"

// Used in level1 - Calculates the error distance of a point for one plane model. Each thread is responsible for one point
__global__ void distance(const float *points, float *output, float m, float c, int N){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){

        // 2 is hard coded but it is 2d points
        float x0 = points[i*2];
        float y0 = points[i*2 + 1];
        // intersection point with the model
        float x1 = (x0 + (m*y0) - (m*c))/(1 + (m*m));
        float y1 = ((m*x0) + ((m*m)*y0) - ((m*m)*c))/(1 + (m*m)) + c;
        float dist = sqrt(((x1 - x0)*(x1 - x0)) + ((y1 - y0)*(y1 - y0)));
        output[i] = dist;
    } 
}

// Given 2 points, find the equation of line passing through it. Each thread is responsible for one line equation
__global__ void find_line_model(const float *maybe_points1, const float *maybe_points2, float *m, float *c, int num_models){

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_models){

        float x1 = maybe_points1[i*2];
        float y1 = maybe_points1[i*2 + 1];

        float x2 = maybe_points2[i*2];
        float y2 = maybe_points2[i*2 + 1];

        m[i] = (y2 - y1)/(x2 - x1 + FLT_MIN);
        c[i] = y2 - m[i]*x2;

    }

}

// Here, error is calculated for all points in all models in parallel. But, the number of points per model cannot exceed 1024.
__global__ void distance_model_parallel(const float *points, float *output, float *m_all, float *c_all, int num_samples){

    int tx = threadIdx.x;
    float m = m_all[blockIdx.x];
    float c = c_all[blockIdx.x];
    int op_idx = blockIdx.x*num_samples + tx;

    if (tx < num_samples){

        float x0 = points[tx*2];
        float y0 = points[tx*2 + 1];

        // intersection point with the model
        float x1 = (x0 + (m*y0) - (m*c))/(1 + (m*m));
        float y1 = ((m*x0) + ((m*m)*y0) - ((m*m)*c))/(1 + (m*m)) + c;
        float dist = sqrt(((x1 - x0)*(x1 - x0)) + ((y1 - y0)*(y1 - y0)));
        output[op_idx] = dist;
    } 
}

// Here, error is calculated for all points in all models in parallel with no limitation on the number of points
__global__ void distance_model_parallel_large(const float *points, float *output, float *m_all, float *c_all, int num_samples){

    int tx = threadIdx.x;
    int point_idx = blockIdx.x * blockDim.x + tx;

    // There are a total of blockDim.y models
    float m = m_all[blockIdx.y];
    float c = c_all[blockIdx.y];

    // The output index for error calculation of one point for one model
    int op_idx = blockIdx.y*num_samples + point_idx;

    if (point_idx < num_samples){

        float x0 = points[point_idx*2];
        float y0 = points[point_idx*2 + 1];

        // intersection point with the model
        float x1 = (x0 + (m*y0) - (m*c))/(1 + (m*m));
        float y1 = ((m*x0) + ((m*m)*y0) - ((m*m)*c))/(1 + (m*m)) + c;
        float dist = sqrt(((x1 - x0)*(x1 - x0)) + ((y1 - y0)*(y1 - y0)));
        output[op_idx] = dist;
    } 
}