
#include <hip/hip_runtime.h>
#include "float.h"
__global__ void distance(const float *points, float *output, float m, float c, int N){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){

        // 2 is hard coded but it is 2d points
        float x0 = points[i*2];
        float y0 = points[i*2 + 1];
        // intersection point with the model
        float x1 = (x0 + (m*y0) - (m*c))/(1 + (m*m));
        float y1 = ((m*x0) + ((m*m)*y0) - ((m*m)*c))/(1 + (m*m)) + c;
        float dist = sqrt(((x1 - x0)*(x1 - x0)) + ((y1 - y0)*(y1 - y0)));
        output[i] = dist;
    } 
}

__global__ void find_line_model(const float *maybe_points1, const float *maybe_points2, float *m, float *c, int num_models){

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_models){

        float x1 = maybe_points1[i*2];
        float y1 = maybe_points1[i*2 + 1];

        float x2 = maybe_points2[i*2];
        float y2 = maybe_points2[i*2 + 1];

        m[i] = (y2 - y1)/(x2 - x1 + FLT_MIN);
        c[i] = y2 - m[i]*x2;

    }

}

__global__ void distance_model_parallel(const float *points, float *output, float *m_all, float *c_all, int num_samples){

    int tx = threadIdx.x;
    float m = m_all[blockIdx.x];
    float c = c_all[blockIdx.x];
    int op_idx = blockIdx.x*num_samples + tx;

    if (tx < num_samples){

        float x0 = points[tx*2];
        float y0 = points[tx*2 + 1];

        // intersection point with the model
        float x1 = (x0 + (m*y0) - (m*c))/(1 + (m*m));
        float y1 = ((m*x0) + ((m*m)*y0) - ((m*m)*c))/(1 + (m*m)) + c;
        float dist = sqrt(((x1 - x0)*(x1 - x0)) + ((y1 - y0)*(y1 - y0)));
        output[op_idx] = dist;
    } 
}

__global__ void distance_model_parallel_large(const float *points, float *output, float *m_all, float *c_all, int num_samples){

    int tx = threadIdx.x;
    int point_idx = blockIdx.x * blockDim.x + tx;

    float m = m_all[blockIdx.y];
    float c = c_all[blockIdx.y];
    int op_idx = blockIdx.y*num_samples + point_idx;

    if (point_idx < num_samples){

        float x0 = points[point_idx*2];
        float y0 = points[point_idx*2 + 1];

        // intersection point with the model
        float x1 = (x0 + (m*y0) - (m*c))/(1 + (m*m));
        float y1 = ((m*x0) + ((m*m)*y0) - ((m*m)*c))/(1 + (m*m)) + c;
        float dist = sqrt(((x1 - x0)*(x1 - x0)) + ((y1 - y0)*(y1 - y0)));
        output[op_idx] = dist;
    } 
}